#include "hip/hip_runtime.h"
#include <algorithm> // std::fill_n
#include <iostream>  // snprintf
#include <math.h>    // expf, log
#include <stdlib.h>  // rand
#include <string>    // std::string
#include <vector>    // std::vector

#include "src/kernels/qkv_bias_and_RoPE.h"
#include "src/weights/llama/attention_weights.h"
#include "src/utils/macro.h"
// (RussWong)note: not sure CPU implementation is absolutely right and the GPU kernel is right compared with HF.
// when you are implementing LLMs inference on CPU, you can reuse the CPU kernel and test its correctness
void CPUfunc(float *q,
             float *k,
             float *v,
             float *QKV,
             const float *qkv_bias,
             const int *padding_offset,
             const int *history_length,
             const int *input_length,
             const int batch_size,
             const int seq_len,
             const int token_num,
             const int head_num,
             const int kv_head_num,
             const int head_size,
             const int rotary_embedding_dim,
             float rotary_embedding_base)
{
    int qbatchstride = seq_len * head_num * head_size;
    int kvbatchstride = seq_len * kv_head_num * head_size;
    for (int b = 0; b < batch_size; b++)
    {
        for (int s = 0; s < seq_len; s++)
        {
            int timestep = history_length[b] + s;
            for (int head = 0; head < head_num; head++)
            {
                for (int d = 0; d < head_size; d++)
                {
                    // q bias
                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d] =
                        QKV[b * qbatchstride + s * head_num * head_size + head * head_size + d];
                }
                // RoPE
                for (int d = 0; d < head_size / 2; d++)
                {
                    float x0 = q[b * qbatchstride + s * head_num * head_size + head * head_size + d];
                    float x1 = q[b * qbatchstride + s * head_num * head_size + head * head_size + d + 64];
                    // refer to https://zhuanlan.zhihu.com/p/647109286, d=0,2,4,dim-1
                    float inv_freq = timestep / powf(rotary_embedding_base, (d * 2) / (float)rotary_embedding_dim);
                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d] =
                        x0 * cos(inv_freq) - x1 * sin(inv_freq);

                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d + 64] =
                        x1 * cos(inv_freq) + x0 * sin(inv_freq);
                }
            }
            for (int head = 0; head < kv_head_num; head++)
            {
                for (int d = 0; d < head_size; d++)
                {
                    // k bias
                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] =
                        QKV[b * kvbatchstride + s * (head_num + kv_head_num) * head_size + head * head_size + d]; // + qkv_bias[(head_num + kv_head_num)  * head_size + d];
                    v[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] =
                        QKV[b * kvbatchstride + s * (head_num + kv_head_num * 2) * head_size + head * head_size + d]; // + qkv_bias[(head_num + 2 * kv_head_num)  * head_size + d];
                }
                // RoPE
                for (int d = 0; d < head_size / 2; d++)
                {
                    float x0 = k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d];
                    float x1 = k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d + 64];
                    float inv_freq = timestep / powf(rotary_embedding_base, (d * 2) / (float)rotary_embedding_dim);
                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] =
                        x0 * cos(inv_freq) - x1 * sin(inv_freq);

                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d + 64] =
                        x1 * cos(inv_freq) + x0 * sin(inv_freq);
                }
            }
        }
    }
}

template <typename T>
bool CheckResult(T *q, T *k, float *hq, float *hk,
                 const int q_size, const int k_size)
{
    for (int i = 0; i < q_size; i++)
    {
        if (fabs((float)q[i] - hq[i]) > 1e-6)
        {
            printf("the %dth q is wrong, q = %f, hq = %f\n", i, q[i], hq[i]);
            return false;
        }
    }
    for (int i = 0; i < k_size; i++)
    {
        if (fabs((float)k[i] - hk[i]) > 1e-6)
        {
            printf("the %dth k is wrong, k = %f, hk = %f\n", i, k[i], hk[i]);
            return false;
        }
    }
    return true;
}

#define TEST_ROPE(dtype)                                                                                                                   \
    dtype *q = (dtype *)malloc(sizeof(dtype) * batch_size * seq_len * head_num * head_size);                                               \
    dtype *k = (dtype *)malloc(sizeof(dtype) * batch_size * seq_len * kv_head_num * head_size);                                            \
    dtype *v = (dtype *)malloc(sizeof(dtype) * batch_size * seq_len * kv_head_num * head_size);                                            \
    dtype *QKV = (dtype *)malloc(sizeof(dtype) * token_num * (head_num + 2 * kv_head_num) * head_size);                                    \
    dtype *qkv_bias = (dtype *)malloc(sizeof(dtype) * (head_num + 2 * kv_head_num) * head_size);                                           \
    for (int i = 0; i < token_num * (head_num + 2 * kv_head_num) * head_size; i++)                                                         \
    {                                                                                                                                      \
        QKV[i] = (dtype)32.0;                                                                                                              \
    }                                                                                                                                      \
    for (int i = 0; i < (head_num + 2 * kv_head_num) * head_size; i++)                                                                     \
    {                                                                                                                                      \
        qkv_bias[i] = (dtype)2.0;                                                                                                          \
    }                                                                                                                                      \
    for (int i = 0; i < batch_size; i++)                                                                                                   \
    {                                                                                                                                      \
        input_length[i] = 7;                                                                                                               \
        history_length[i] = 0;                                                                                                             \
    }                                                                                                                                      \
    for (int i = 0; i < batch_size * seq_len; i++)                                                                                         \
    {                                                                                                                                      \
        padding_offset[i] = 0;                                                                                                             \
    }                                                                                                                                      \
    int *dpadding_offset;                                                                                                                  \
    int *dhistory_length;                                                                                                                  \
    int *dinput_length;                                                                                                                    \
    dtype *dq;                                                                                                                             \
    dtype *dk;                                                                                                                             \
    dtype *dv;                                                                                                                             \
    dtype *dQKV;                                                                                                                           \
    dtype *dqkv_bias;                                                                                                                      \
    hipMalloc((void **)&dpadding_offset, sizeof(int) * batch_size * seq_len);                                                             \
    hipMalloc((void **)&dhistory_length, sizeof(int) * batch_size);                                                                       \
    hipMalloc((void **)&dinput_length, sizeof(int) * batch_size);                                                                         \
    hipMalloc((void **)&dq, sizeof(dtype) * batch_size * seq_len * head_num * head_size);                                                 \
    hipMalloc((void **)&dk, sizeof(dtype) * batch_size * seq_len * kv_head_num * head_size);                                              \
    hipMalloc((void **)&dv, sizeof(dtype) * batch_size * seq_len * kv_head_num * head_size);                                              \
    hipMalloc((void **)&dQKV, sizeof(dtype) * token_num * (head_num + 2 * kv_head_num) * head_size);                                      \
    hipMalloc((void **)&dqkv_bias, sizeof(dtype) * (head_num + 2 * kv_head_num) * head_size);                                             \
    hipMemcpy(dinput_length, input_length, sizeof(int) * batch_size, hipMemcpyHostToDevice);                                             \
    hipMemcpy(dhistory_length, history_length, sizeof(int) * batch_size, hipMemcpyHostToDevice);                                         \
    hipMemcpy(dpadding_offset, padding_offset, sizeof(int) * seq_len * batch_size, hipMemcpyHostToDevice);                               \
    hipMemcpy(dQKV, QKV, sizeof(dtype) * token_num * (head_num + 2 * kv_head_num) * head_size, hipMemcpyHostToDevice);                   \
    hipMemcpy(dqkv_bias, qkv_bias, sizeof(dtype) * (head_num + 2 * kv_head_num) * head_size, hipMemcpyHostToDevice);                     \
    DataType type = getTensorType<dtype>();                                                                                                \
    TensorWrapper<dtype> *q_buf = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, head_num, seq_len, head_size}, dq);             \
    TensorWrapper<dtype> *k_buf = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, kv_head_num, seq_len, head_size}, dk);          \
    TensorWrapper<dtype> *v_buf = new TensorWrapper<dtype>(Device::GPU, type, {batch_size, kv_head_num, seq_len, head_size}, dv);          \
    TensorWrapper<dtype> *QKV_buf = new TensorWrapper<dtype>(Device::GPU, type, {token_num, head_num + 2 * kv_head_num, head_size}, dQKV); \
    LLaMAattentionWeights<dtype> attn_weights;                                                                                             \
    attn_weights.qkv.bias = dqkv_bias;                                                                                                     \
    DataType type_int = getTensorType<int>();                                                                                              \
    TensorWrapper<int> *input_length_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, dinput_length);                     \
    TensorWrapper<int> *history_length_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, dhistory_length);                 \
    TensorWrapper<int> *padding_offset_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size, seq_len}, dpadding_offset);        \
    LLaMAAttentionStaticParams params;                                                                                                     \
    params.rotary_embedding_dim = rotary_embedding_dim;                                                                                    \
    params.rotary_embedding_base = rotary_embedding_base;                                                                                  \
    params.max_position_embeddings = max_position_embeddings;                                                                              \
    params.use_dynamic_ntk = false;                                                                                                        \
    std::cout << "before launch kernel" << std::endl;                                                                                      \
    launchAddFusedQKVBiasTransposeAndRoPE(q_buf,                                                                                           \
                                          k_buf,                                                                                           \
                                          v_buf,                                                                                           \
                                          QKV_buf,                                                                                         \
                                          attn_weights.qkv,                                                                                \
                                          padding_offset_buf,                                                                              \
                                          history_length_buf,                                                                              \
                                          input_length_buf,                                                                                \
                                          params);                                                                                         \
    std::cout << "after launch kernel" << std::endl;                                                                                       \
    std::cout << "cuda memcpy device to host" << std::endl;                                                                                \
    CHECK(hipMemcpy(q, dq, sizeof(dtype) * batch_size * seq_len * head_num * head_size, hipMemcpyDeviceToHost));                         \
    CHECK(hipMemcpy(k, dk, sizeof(dtype) * batch_size * seq_len * kv_head_num * head_size, hipMemcpyDeviceToHost));                      \
    std::cout << "after memcpyd2h, dq[0] = " << (float)q[0] << std::endl;                                                                         \
    std::cout << "before CPU function" << std::endl;                                                                                       \
    float *hq = (float *)malloc(sizeof(float) * batch_size * seq_len * head_num * head_size);                                              \
    float *hk = (float *)malloc(sizeof(float) * batch_size * seq_len * kv_head_num * head_size);                                           \
    float *hv = (float *)malloc(sizeof(float) * batch_size * seq_len * kv_head_num * head_size);                                           \
    float *hQKV = (float *)malloc(sizeof(float) * token_num * (head_num + 2 * kv_head_num) * head_size);                                   \
    float *hqkv_bias = (float *)malloc(sizeof(float) * (head_num + 2 * kv_head_num) * head_size);                                          \
    for (int i = 0; i < token_num * (head_num + 2 * kv_head_num) * head_size; i++)                                                         \
    {                                                                                                                                      \
        hQKV[i] = 32.0f;                                                                                                                   \
    }                                                                                                                                      \
    for (int i = 0; i < (head_num + 2 * kv_head_num) * head_size; i++)                                                                     \
    {                                                                                                                                      \
        hqkv_bias[i] = 2.0f;                                                                                                               \
    }                                                                                                                                      \
    CPUfunc(hq,                                                                                                                            \
            hk,                                                                                                                            \
            hv,                                                                                                                            \
            hQKV,                                                                                                                          \
            hqkv_bias,                                                                                                                     \
            padding_offset,                                                                                                                \
            history_length,                                                                                                                \
            input_length,                                                                                                                  \
            batch_size,                                                                                                                    \
            seq_len,                                                                                                                       \
            token_num,                                                                                                                     \
            head_num,                                                                                                                      \
            kv_head_num,                                                                                                                   \
            head_size,                                                                                                                     \
            rotary_embedding_dim,                                                                                                          \
            rotary_embedding_base);                                                                                                        \
    std::cout << "after CPU function" << std::endl;                                                                                        \
    bool is_right = CheckResult<dtype>(q, k, hq, hk,                                                                                       \
                                       batch_size * seq_len * head_num * head_size,                                                        \
                                       batch_size * seq_len * kv_head_num * head_size);                                                    \
    std::cout << "before free" << std::endl;                                                                                               \
    std::cout << "passed" << std::endl;                                                                                                    \
    free(q);                                                                                                                               \
    free(k);                                                                                                                               \
    free(v);                                                                                                                               \
    free(QKV);                                                                                                                             \
    free(qkv_bias);                                                                                                                        \
    free(padding_offset);                                                                                                                  \
    free(history_length);                                                                                                                  \
    free(input_length);                                                                                                                    \
    free(hq);                                                                                                                              \
    free(hk);                                                                                                                              \
    free(hv);                                                                                                                              \
    free(hQKV);                                                                                                                            \
    free(hqkv_bias);                                                                                                                       \
    hipFree(dq);                                                                                                                          \
    hipFree(dk);                                                                                                                          \
    hipFree(dv);                                                                                                                          \
    hipFree(dQKV);                                                                                                                        \
    hipFree(dqkv_bias);                                                                                                                   \
    hipFree(dpadding_offset);                                                                                                             \
    hipFree(dhistory_length);                                                                                                             \
    hipFree(dinput_length);

// (RussWong)note:
// `./biasRope` to test fp32 GPU kernel
// half GPU kernel test is not implemented now
int main(int argc, char *argv[])
{
    const int batch_size = 1;
    const int seq_len = 32;
    int *padding_offset = (int *)malloc(sizeof(int) * batch_size * seq_len);
    int *history_length = (int *)malloc(sizeof(int) * batch_size);
    int *input_length = (int *)malloc(sizeof(int) * batch_size);
    const int token_num = batch_size * seq_len;
    const int head_num = 32;
    const int kv_head_num = 32;
    const int head_size = 128;
    const int rotary_embedding_dim = 128;
    const int rotary_embedding_base = 10000;
    const int max_position_embeddings = 2048;
    if (argv[1])
    {
        TEST_ROPE(half);
    } else {
        TEST_ROPE(float);
    }
}
